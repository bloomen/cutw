#include "copy.h"

#include <hip/hip_runtime.h>

#include "error.h"

namespace cutw
{

namespace detail
{

void copy_to_device_impl(const void* const host, void* const device,
                         const std::size_t size, Stream* const s)
{
    if (s)
    {
        CUTW_CUASSERT(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice, s->get()));
    }
    else
    {
        CUTW_CUASSERT(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
    }
}

void copy_to_host_impl(void* const host, const void* const device,
                       const std::size_t size, Stream* const s)
{
    if (s)
    {
        CUTW_CUASSERT(hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost, s->get()));
    }
    else
    {
        CUTW_CUASSERT(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }
}

}

}
