#include "DeviceArray.h"

#include <hip/hip_runtime.h>

#include "error.h"

namespace cutw
{

namespace detail
{

void device_allocate(void*& data, const std::size_t bytes)
{
    CUTW_CUASSERT(hipMalloc(&data, bytes));
}

void device_free(void* const data)
{
    CUTW_CUASSERT(hipFree(data));
}

}

}
