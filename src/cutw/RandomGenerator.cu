#include "RandomGenerator.h"

#include <hiprand/hiprand_kernel.h>

#include "error.h"

namespace cutw
{

struct RandomGenerator::impl
{
    impl()
    {
        CUTW_CURANDASSERT(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    }
    ~impl()
    {
        CUTW_CURANDASSERT(hiprandDestroyGenerator(gen));
    }
    hiprandGenerator_t gen;
};

RandomGenerator::RandomGenerator(Stream& stream, const std::size_t seed)
    : impl_{new impl}
{
    CUTW_CURANDASSERT(hiprandSetPseudoRandomGeneratorSeed(impl_->gen, seed));
    CUTW_CURANDASSERT(hiprandSetStream(impl_->gen, stream.get()));
}

RandomGenerator::~RandomGenerator()
{}

void RandomGenerator::generateUniform(float* const device, const std::size_t n)
{
    CUTW_CURANDASSERT(hiprandGenerateUniform(impl_->gen, device, n));
}

void RandomGenerator::generateUniform(double* const device, const std::size_t n)
{
    CUTW_CURANDASSERT(hiprandGenerateUniformDouble(impl_->gen, device, n));
}

hiprandGenerator_t RandomGenerator::get() const
{
    return impl_->gen;
}

}
