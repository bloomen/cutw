#include "RandomGenerator.h"

#include <hiprand/hiprand_kernel.h>

#include "error.h"

namespace cutw
{

struct RandomGenerator::impl
{
    impl()
    {
        CUTW_CURANDASSERT(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    }
    ~impl()
    {
        CUTW_CURANDASSERT(hiprandDestroyGenerator(gen));
    }
    hiprandGenerator_t gen;
};

RandomGenerator::RandomGenerator(const std::size_t seed)
    : impl_{new impl}
{
    CUTW_CURANDASSERT(hiprandSetPseudoRandomGeneratorSeed(impl_->gen, seed));
}

RandomGenerator::~RandomGenerator()
{}

void RandomGenerator::generateUniform(Stream& stream, float* const device, const std::size_t n)
{
    CUTW_CURANDASSERT(hiprandSetStream(impl_->gen, stream.get()));
    CUTW_CURANDASSERT(hiprandGenerateUniform(impl_->gen, device, n));
}

void RandomGenerator::generateUniform(Stream& stream, double* const device, const std::size_t n)
{
    CUTW_CURANDASSERT(hiprandSetStream(impl_->gen, stream.get()));
    CUTW_CURANDASSERT(hiprandGenerateUniformDouble(impl_->gen, device, n));
}

hiprandGenerator_t RandomGenerator::get() const
{
    return impl_->gen;
}

}
