#include "Stream.h"

#include <hip/hip_runtime.h>

#include "error.h"

namespace cutw
{

struct Stream::impl
{
    impl()
    {
        CUTW_CUASSERT(hipStreamCreate(&stream));
    }
    ~impl()
    {
        CUTW_CUASSERT(hipStreamDestroy(stream));
    }
    hipStream_t stream;
};

Stream::Stream()
    : impl_{new impl}
{}

Stream::~Stream()
{}

hipStream_t Stream::get() const
{
    return impl_->stream;
}

void Stream::sync()
{
    CUTW_CUASSERT(hipStreamSynchronize(impl_->stream));
}

}
