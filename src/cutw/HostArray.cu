#include "HostArray.h"

#include <hip/hip_runtime.h>

#include "error.h"

namespace cutw
{

namespace detail
{

void host_allocate(void*& data, const std::size_t bytes)
{
    CUTW_CUASSERT(hipHostMalloc(&data, bytes));
}

void host_free(void* const data)
{
    CUTW_CUASSERT(hipHostFree(data));
}

}

}
